#include "hip/hip_runtime.h"
#include <torch/extension.h>
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>
#include <>
#include <corecrt_math_defines.h>

#define N 8

__global__
void
fft_naive(hipFloatComplex* x, int n, int steps){
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	for (int stage = 0; stage < steps; ++stage){
		int numOfElem = 1 << (stage + 1);
		int xStart = idx * numOfElem;
		if (xStart < n){
			for(int k = 0; k < numOfElem/2; ++k){
				// Butterfly Operation
				float angle = -2.0f * M_PI * k / numOfElem;
				hipFloatComplex rotation = make_hipFloatComplex(cos(angle), sin(angle));

				hipFloatComplex even = x[xStart+k];
				hipFloatComplex odd = x[xStart + numOfElem/2 + k];
				hipFloatComplex twiddle = hipCmulf(rotation,odd);
				x[xStart+k] = hipCaddf(even,twiddle);
				x[xStart + numOfElem/2 + k] = hipCsubf(even, twiddle);  
			}
		}
		__syncthreads();
	}
}

/**
 * @brief reverse the given number's bit representation
 * 
 * @param num 	the num to be processed
 * @param bits 	the width to be processed 
 * @return int 	the reversed num
 */
int
reverse_bit(int num, int bits){
	int res = 0;
	for (int i = 0; i < bits; ++i){
		res = (res << 1) | (num & 1);
		num >>= 1;
	}
	return res;
}

/**
 * @brief 	bit-reverse a given array
 * 
 * @param arr 
 * @param n	length of the array
 * @return int status, 0 for success
 */
int 
bit_reverse(hipFloatComplex* arr, int n){
	// get the bits needed to represent the length
	int bits = 0;
	while ((1 << bits) < n){
		bits++;
	}
	// swap the reverse-bits pair
	for (int i = 0; i < n; ++i){
		int j = reverse_bit(i,bits);
		if (i < j){
			hipFloatComplex temp = arr[i];
			arr[i] = arr[j];
			arr[j] = temp;
		}
	}
	return 0;
}

/**
 * @brief 
 * 
 * @param res 
 * @param n 
 * @return int 
 */
int print_res(hipFloatComplex* res, int n){
	printf("Index\tValue\n");
	for(int i = 0; i < n; ++i){
		printf("[%d]:\t(%.2f, %.2f)\n", i, res[i].x, res[i].y);
	}
	printf("\n");
	return 0;
}

int fft(hipFloatComplex* x_h, int n){
	hipFloatComplex* x_d;
	// Allocate and Copy
	int sz = N * sizeof hipFloatComplex;
	hipMalloc((void**)&x_d, sz);
	hipMemcpy(x_d, x_h,sz,hipMemcpyHostToDevice);	
	// Launch the Kernel
	fft_naive<<<1,512>>>(x_d, n, log2(n));
	// Post process
	hipMemcpy(x_h,x_d,sz,hipMemcpyDeviceToHost);
	hipFree(x_d);
	return 0;
}

/**
 * @brief 
 * 
 * @param argc 
 * @param argv 
 * @return int 
 */
int main(int argc, char* argv[]) {
	
	hipFloatComplex *x_h;
	x_h = (hipFloatComplex*) malloc(N * sizeof hipFloatComplex);
	// initialize 
	for (int i = 0; i < N; ++i){
		x_h[i] = make_hipFloatComplex((float)i,0.0);
	}
	print_res(x_h, N);
	// Do Bit Reverse in the host
	bit_reverse(x_h,N);
	fft(x_h, N);
	print_res(x_h,N);
	// free host memory
	free(x_h);
	return 0;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("fft_naive", &fft_naive, "FFT naive implementation(CUDA)");
}